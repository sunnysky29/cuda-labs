#include "hip/hip_runtime.h"
/*

以看到，矩阵C中各个元素的计算是独立的。可以利用GPU的大量线程去分别计算矩阵C
 中相应的元素，以达到并行计算的目的，
*/


#include <omp.h>
#include <Eigen/Core>
#include <ctime>
#include <iostream>

__global__ void gemmKernel(const float *A, const float *B, float *C,
                          float alpha, float beta, unsigned M, unsigned N,
                          unsigned K) {
    unsigned int m = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int n = threadIdx.y + blockDim.y * blockIdx.y;
    if (m >= M || n >= N) return;
    float c = 0;
    for (unsigned k = 0; k < K; ++k) {
        c += A[m * K + k] * B[k * N + n];
    }
    c = c * alpha;
    float result = c;
    if (beta != 0) {
        result = result + C[m * N + n] * beta;
    }
    C[m * N + n] = result;
}

void gemmNaive(const float *A, const float *B, float *C, float alpha,
               float beta, unsigned M, unsigned N, unsigned K) {
    // dim3 block(32, 32);
    dim3 block(16,16);

    dim3 grid((M - 1) / block.x + 1, (N - 1) / block.y + 1);
    gemmKernel<<<grid, block>>>(A, B, C, alpha, beta, M, N, K);
}

int main() {
    // GPU信息输出
    int gpu_rank = 0;
    hipDeviceProp_t deviceProp{};
    hipGetDeviceProperties(&deviceProp, gpu_rank);
    hipSetDevice(gpu_rank);
    printf("GPU %s status: ", deviceProp.name);
    double boostFrequency = deviceProp.clockRate / 1e6;
    int fp32CoresNum = 640;
    double peakPerformance = boostFrequency * fp32CoresNum * 2;
    printf("clock rate %.3f GHz, FP32 cores num %d, FP32 peak throughput %.3f GFLOPS\n",
           boostFrequency, fp32CoresNum, peakPerformance);

    // 设置矩阵大小和参数
    omp_set_num_threads(omp_get_num_procs());
    unsigned M = 1024, N = 1024, K = 1024;
    float alpha = 1., beta = 0.;
    
    std::srand(static_cast<unsigned int>(std::time(nullptr))); // 使用当前时间作为种子

    // 分配设备内存
    float *deviceAPtr, *deviceBPtr, *deviceCPtr;
    Eigen::Matrix<float, Eigen::Dynamic, Eigen::Dynamic, Eigen::RowMajor> A{M, K}, B{K, N}, C{M, N};
    
    // 初始化随机矩阵
    A.setRandom();
    B.setRandom();
    C.setRandom();
    
    // 打印第一个元素作为示例
    std::cout << "Matrix A first element: " << A(0, 0) << std::endl;
    std::cout << "Matrix B first element: " << B(0, 0) << std::endl;
    std::cout << "Matrix C first element: " << C(0, 0) << std::endl;

    // 拷贝数据到设备
    hipMalloc(&deviceAPtr, M * K * sizeof(float));
    hipMemcpy(deviceAPtr, A.data(), M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&deviceBPtr, K * N * sizeof(float));
    hipMemcpy(deviceBPtr, B.data(), K * N * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&deviceCPtr, M * N * sizeof(float));
    hipMemcpy(deviceCPtr, C.data(), M * N * sizeof(float), hipMemcpyHostToDevice);

    // 计时GPU计算
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipEventRecord(startEvent);
    gemmNaive(deviceAPtr, deviceBPtr, deviceCPtr, alpha, beta, M, N, K);
    hipEventRecord(stopEvent);
    hipEventSynchronize(stopEvent);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, startEvent, stopEvent);
    printf("GPU time: %.3f ms\n", milliseconds);
    hipEventDestroy(stopEvent);
    hipEventDestroy(startEvent);

    // CPU计算
    Eigen::Matrix<float, Eigen::Dynamic, Eigen::Dynamic, Eigen::RowMajor> hostResult{M, N}, deviceResult{M, N};
    clock_t begin, end;
    begin = clock();
    hostResult = alpha * (A * B) + beta * C;
    end = clock();
    printf("CPU time: %.3f ms\n", double(end - begin) / CLOCKS_PER_SEC * 1e3);

    // 拷贝结果回主机
    hipMemcpy(deviceResult.data(), deviceCPtr, M * N * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // 打印第一个结果元素
    std::cout << "CPU result first element: " << hostResult(0, 0) << std::endl;
    std::cout << "GPU result first element: " << deviceResult(0, 0) << std::endl;

    // 计算误差
    Eigen::Array<float, Eigen::Dynamic, Eigen::Dynamic> diffArray = (hostResult - deviceResult).array().abs();
    printf("Max Error: %f\n", diffArray.maxCoeff());

    // 计算GFLOPS
    double GFLOPS = 2 * 1e-9 * M * N * K / (milliseconds * 1e-3);
    printf("GPU Throughput: %.3f GFLOPS\n", GFLOPS);

    // 释放设备内存
    hipFree(deviceAPtr);
    hipFree(deviceBPtr);
    hipFree(deviceCPtr);

    return 0;
}