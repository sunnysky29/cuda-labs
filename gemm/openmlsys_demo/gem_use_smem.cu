#include "hip/hip_runtime.h"
#include "util.cuh"
#include <iostream>
#include <ctime>
#include <cstdlib>
#include <omp.h>
#include <Eigen/Core>


namespace {
template <typename LayoutTile, typename LayoutBlock, typename LayoutThread>
__global__ void gemmKernel(const float *__restrict__ A,
                           const float *__restrict__ B, float *__restrict__ C,
                           float alpha, float beta, unsigned M, unsigned N,
                           unsigned K) {
  constexpr unsigned ratio = sizeof(openmlsys::float4) / sizeof(float);
  using LayoutTileT =
      openmlsys::Layout<LayoutTile::m / ratio, LayoutTile::n / ratio,
                        LayoutTile::k / ratio>;
  using LayoutThreadT =
      openmlsys::Layout<LayoutThread::m / ratio, LayoutThread::n / ratio>;
  constexpr unsigned blockSize = LayoutBlock::m * LayoutBlock::n;
  constexpr openmlsys::float4 float4Zero{0.f, 0.f, 0.f, 0.f};

  __shared__ openmlsys::float4 tileA[LayoutTile::m][LayoutTileT::k];
  __shared__ openmlsys::float4 tileB[LayoutTile::k][LayoutTileT::n];

  const unsigned nInTileC = threadIdx.x % LayoutBlock::m;
  const unsigned mInTileC = threadIdx.x / LayoutBlock::m;

  const unsigned kInTileA = threadIdx.x % LayoutTileT::k;
  const unsigned mInTileA = threadIdx.x / LayoutTileT::k;

  const unsigned nInTileB = threadIdx.x % LayoutTileT::n;
  const unsigned kinTileB = threadIdx.x / LayoutTileT::n;

  openmlsys::Tensor2D<const openmlsys::float4> pA{A, M, K / ratio};
  pA.addOffset(LayoutTile::m * blockIdx.y + mInTileA, kInTileA);
  openmlsys::Tensor2D<const openmlsys::float4> pB{B, K, N / ratio};
  pB.addOffset(kinTileB,
               LayoutTileT::n * blockIdx.x + nInTileB * LayoutThreadT::n);
  openmlsys::Tensor2D<openmlsys::float4> pC{C, M, N / ratio};
  pC.addOffset(LayoutTile::m * blockIdx.y + mInTileC * LayoutThread::m,
               LayoutTileT::n * blockIdx.x + nInTileC * LayoutThreadT::n);

  constexpr unsigned tileSizeA = LayoutTile::m * LayoutTile::k;
  constexpr unsigned tileSizeB = LayoutTile::n * LayoutTile::k;
  constexpr unsigned tileIterationsA = tileSizeA / blockSize / ratio;
  constexpr unsigned tileGlobalIntervalA = blockSize / LayoutTileT::k;
  constexpr unsigned tileComputeIterationsA = LayoutTileT::m / LayoutBlock::m;
  constexpr unsigned tileSharedIntervalA =
      LayoutTile::m / tileComputeIterationsA;
  constexpr unsigned tileIterationsB = tileSizeB / blockSize / ratio;
  constexpr unsigned tileGlobalIntervalB = blockSize / LayoutTileT::n;
  constexpr unsigned tileComputeIterationsB = LayoutTileT::n / LayoutBlock::n;
  constexpr unsigned tileSharedIntervalBT =
      LayoutTileT::n / tileComputeIterationsB;

  openmlsys::float4 bufferA[tileIterationsA];
  openmlsys::float4 bufferB[tileIterationsB];
  bool validLoadTileA[tileIterationsA];
  bool validLoadTileB[tileIterationsB];

#pragma unroll
  for (unsigned i = 0; i < tileIterationsA; ++i) {
    validLoadTileA[i] = pA.validRowOffset(i * tileGlobalIntervalA);
  }

#pragma unroll
  for (unsigned i = 0; i < tileIterationsB; ++i) {
    validLoadTileB[i] = pB.validColOffset(0);
  }

  openmlsys::float4 c[tileComputeIterationsA * LayoutThread::m]
                     [tileComputeIterationsB * LayoutThreadT::n];
  memset(c, 0, sizeof(c));

  openmlsys::float4 fragmentA[tileComputeIterationsA * LayoutThreadT::m];
  openmlsys::float4 fragmentB[tileComputeIterationsB * LayoutThreadT::n];

  for (unsigned i = 0; i < K; i += LayoutTile::k) {
#pragma unroll
    for (unsigned j = 0; j < tileIterationsA; ++j) {
      validLoadTileA[j] &= pA.validColOffset(0);
      bufferA[j] =
          validLoadTileA[j] ? pA(j * tileGlobalIntervalA, 0) : float4Zero;
    }

#pragma unroll
    for (unsigned j = 0; j < tileIterationsB; ++j) {
      validLoadTileB[j] &= pB.validRowOffset(j * tileGlobalIntervalB);
      bufferB[j] =
          validLoadTileB[j] ? pB(j * tileGlobalIntervalB, 0) : float4Zero;
    }

    __syncthreads();
#pragma unroll
    for (unsigned a = 0; a < tileIterationsA; ++a) {
      tileA[mInTileA + a * tileGlobalIntervalA][kInTileA] = bufferA[a];
    }

#pragma unroll
    for (unsigned a = 0; a < tileIterationsB; ++a) {
      tileB[kinTileB + a * tileGlobalIntervalB][nInTileB] = bufferB[a];
    }
    __syncthreads();

#pragma unroll
    for (unsigned j = 0; j < LayoutTile::k; j++) {
#pragma unroll
      for (unsigned a = 0; a < tileComputeIterationsA; ++a) {
#pragma unroll
        for (unsigned b = 0; b < LayoutThread::m; ++b) {
          fragmentA[a][b] =
              tileA[a * tileSharedIntervalA + mInTileC * LayoutThread::m + b]
                   [j / ratio][j % ratio];
        }
      }
#pragma unroll
      for (unsigned a = 0; a < tileComputeIterationsB; ++a) {
        fragmentB[a] = tileB[j][a * tileSharedIntervalBT + nInTileC];
      }
#pragma unroll
      for (unsigned d = 0; d < tileComputeIterationsA * LayoutThread::m; ++d) {
#pragma unroll
        for (unsigned e = 0; e < tileComputeIterationsB * LayoutThreadT::n;
             ++e) {
          c[d][e] =
              c[d][e] + fragmentB[e] *
                            fragmentA[d / LayoutThread::m][d % LayoutThread::m];
        }
      }
    }
    pA.addOffset(0, LayoutTileT::k);
    pB.addOffset(LayoutTile::k, 0);
  }

#pragma unroll
  for (auto &a : c) {
#pragma unroll
    for (auto &b : a) {
      b = b * alpha;
    }
  }

#pragma unroll
  for (unsigned i = 0; i < tileComputeIterationsA; ++i) {
#pragma unroll
    for (unsigned a = 0; a < LayoutThread::m; a++) {
      const bool mValid = pC.validRowOffset(a);
#pragma unroll
      for (unsigned b = 0; b < tileComputeIterationsB; b++) {
        const bool nValid = pC.validColOffset(b * tileSharedIntervalBT);
        if (mValid && nValid) {
          openmlsys::float4 result{c[a + i * LayoutThread::m][b]};
          if (beta != 0) {
            result = result + pC(a, b * tileSharedIntervalBT) * beta;
          }
          pC(a, b * tileSharedIntervalBT) = result;
        }
      }
    }
    pC.addOffset(tileSharedIntervalA, 0);
  }
}
}  // namespace

void gemmUseSmem(const float *deviceAPtr, const float *deviceBPtr,
                 float *deviceCPtr, float alpha, float beta, unsigned M,
                 unsigned N, unsigned K) {
  using LayoutTile = openmlsys::Layout<128, 128, 16>;
  using LayoutBlock = openmlsys::Layout<16, 16>;
  using LayoutThread = openmlsys::Layout<4, 4>;

  dim3 block(LayoutBlock::m * LayoutBlock::n);
  dim3 grid((M - 1) / LayoutTile::m + 1, (N - 1) / LayoutTile::n + 1);

  gemmKernel<LayoutTile, LayoutBlock, LayoutThread><<<grid, block>>>(
      deviceAPtr, deviceBPtr, deviceCPtr, alpha, beta, M, N, K);
}


int main() {
    // GPU信息输出
    int gpu_rank = 0;
    hipDeviceProp_t deviceProp{};
    hipGetDeviceProperties(&deviceProp, gpu_rank);
    hipSetDevice(gpu_rank);
    printf("GPU %s status: ", deviceProp.name);
    double boostFrequency = deviceProp.clockRate / 1e6;
    int fp32CoresNum = 640;
    double peakPerformance = boostFrequency * fp32CoresNum * 2;
    printf("clock rate %.3f GHz, FP32 cores num %d, FP32 peak throughput %.3f GFLOPS\n",
           boostFrequency, fp32CoresNum, peakPerformance);
  
    // 设置矩阵大小和参数
    omp_set_num_threads(omp_get_num_procs());
    unsigned M = 1024, N = 1024, K = 1024;
    float alpha = 1., beta = 0.;
    
    std::srand(static_cast<unsigned int>(std::time(nullptr))); // 使用当前时间作为种子
  
    // 分配设备内存
    float *deviceAPtr, *deviceBPtr, *deviceCPtr;
    Eigen::Matrix<float, Eigen::Dynamic, Eigen::Dynamic, Eigen::RowMajor> A{M, K}, B{K, N}, C{M, N};
    
    // 初始化随机矩阵
    A.setRandom();
    B.setRandom();
    C.setRandom();
    
    // 打印第一个元素作为示例
    std::cout << "Matrix A first element: " << A(0, 0) << std::endl;
    std::cout << "Matrix B first element: " << B(0, 0) << std::endl;
    std::cout << "Matrix C first element: " << C(0, 0) << std::endl;
  
    // 拷贝数据到设备
    hipMalloc(&deviceAPtr, M * K * sizeof(float));
    hipMemcpy(deviceAPtr, A.data(), M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&deviceBPtr, K * N * sizeof(float));
    hipMemcpy(deviceBPtr, B.data(), K * N * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&deviceCPtr, M * N * sizeof(float));
    hipMemcpy(deviceCPtr, C.data(), M * N * sizeof(float), hipMemcpyHostToDevice);
  
    // 计时GPU计算
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipEventRecord(startEvent);
    gemmUseSmem(deviceAPtr, deviceBPtr, deviceCPtr, alpha, beta, M, N, K);
    hipEventRecord(stopEvent);
    hipEventSynchronize(stopEvent);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, startEvent, stopEvent);
    printf("GPU time: %.3f ms\n", milliseconds);
    hipEventDestroy(stopEvent);
    hipEventDestroy(startEvent);
  
    // CPU计算
    Eigen::Matrix<float, Eigen::Dynamic, Eigen::Dynamic, Eigen::RowMajor> hostResult{M, N}, deviceResult{M, N};
    clock_t begin, end;
    begin = clock();
    hostResult = alpha * (A * B) + beta * C;
    end = clock();
    printf("CPU time: %.3f ms\n", double(end - begin) / CLOCKS_PER_SEC * 1e3);
  
    // 拷贝结果回主机
    hipMemcpy(deviceResult.data(), deviceCPtr, M * N * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
  
    // 打印第一个结果元素
    std::cout << "CPU result first element: " << hostResult(0, 0) << std::endl;
    std::cout << "GPU result first element: " << deviceResult(0, 0) << std::endl;
  
    // 计算误差
    Eigen::Array<float, Eigen::Dynamic, Eigen::Dynamic> diffArray = (hostResult - deviceResult).array().abs();
    printf("Max Error: %f\n", diffArray.maxCoeff());
  
    // 计算GFLOPS
    double GFLOPS = 2 * 1e-9 * M * N * K / (milliseconds * 1e-3);
    printf("GPU Throughput: %.3f GFLOPS\n", GFLOPS);
  
    // 释放设备内存
    hipFree(deviceAPtr);
    hipFree(deviceBPtr);
    hipFree(deviceCPtr);
  
    return 0;
  }